#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <time.h>
#include <sys/time.h>
#include <string>
#include <random>
#include <algorithm>
#include <functional>

#include <gtest/gtest.h>
#include <utilities/legacy/error_utils.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>

#include <ogrsf_frmts.h>
#include <geos_c.h>


#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>

#include <utility/utility.hpp>
#include <utility/quadtree_thrust.cuh>
#include <utility/bbox_thrust.cuh>

#include <cuspatial/quadtree.hpp>
#include <cuspatial/polygon_bbox.hpp>
#include <cuspatial/spatial_jion.hpp>

#include "spatial_join_test_utility.hpp"

/*
* Test code for running spatial join on GPUs with GDAL/OGR-based CPU verification.
* Different from spatial_join_refinement_small and spatial_join_refinement_large,
* where the expected results can be either embedded in code or computed on CPU (using GDAL/OGR API),
* for yearly NYC taxitrip data, that would take donzes of hours or even more. 
* A sampling-based verificaiton is thus needed. 
*
* Two sampling strategies are provided: sampling on points and sampling on quadrant-polygon pairs.
*
* Furthermore, sophisticated polygons such as multi-polgyons and polygons with holes, 
* are challenging for verfication/debugging purposes .
* This test code allows picking up two types of polygons, i.e., single-ring (classical) 
* and multi-ring polygons and their combinations (all). 
*
* The comparison/vericiation code is also accelerated by using a composition of 
* lower_bound/upper_bound/binary_search paralell primiitves in Thrust. 
* Searching std::vector on CPU is just too slow for this prurpose.   

* As the relationship between points and polygons is many-to-many, the verification gives three metrics: 
* num_search_pnt: numbers of points (from both sampling strategies) that are within at least
* one polygons by CPU code; Disagreement between num_search_pnt and num_pp_pairs indciate mismatches;
* num_not_found: # of point indices of GDAL/OGR CPU results can not be found in GPU results
* num_mis_match: for the same point index,if its assoicated non-empty polygon sets are different 
* between CPU and GPU results, num_mis_match will be increased by 1.

* For a perfert agreement between CPU and GPU results, 
* (num_search_pnt==num_pp_pairs && num_not_found==0 && num_mis_match==0);
*
* naming convention: *_pnt_* (points), *_poly_* (polygon), *_quad_* (quadrant) 
* naming convention: *_qt_* (quadtree), *_pq_* (polygon-quadrant pair), *_pp_* (polygon-point pair) 
* naming convention: h_*(host vairable), d_*(device variable), *_vec (std::vector), *_idx_ (index/offset)
*/

struct SpatialJoinNYCTaxiVerify : public GdfTest 
{        
    uint32_t num_pnts=0;

    uint32_t num_quadrants=0;

    uint32_t num_pq_pairs=0;

    uint32_t num_pp_pairs=0;
   
     //point x/y on host
    double *h_pnt_x=nullptr,*h_pnt_y=nullptr;

    uint32_t num_poly=0,num_ring=0,num_vertex=0;

    //polygon vertices x/y
    double *h_poly_x=nullptr,*h_poly_y=nullptr;

    //quadtree length/fpos
    uint32_t *h_qt_length=nullptr,*h_qt_fpos=nullptr;   

    //quadrant/polygon pairs
    uint32_t *h_pq_quad_idx=nullptr,*h_pq_poly_idx=nullptr;   
    
    uint32_t *h_pp_pnt_idx=nullptr,*h_pp_poly_idx=nullptr;

    //poygons using GDAL/OGR OGRGeometry structure
    std::vector<OGRGeometry *> h_ogr_polygon_vec;
    std::vector<GEOSGeometry *> h_geos_polygon_vec;

    //sequential idx 0..num_poly-1 to index h_ogr_polygon_vec
    //needed when actual polygons in spatial join are only a subset, e.g., multi-polygons only  
    std::vector<uint32_t> h_org_poly_idx_vec;

    //point idx that intersect with at least one polygon based on GDAL/OGR OGRGeometry.Contains 
    std::vector<uint32_t> h_pnt_idx_vec;
    
    //# of poylgons that are contain points indexed by h_pnt_idx_vec at the same index
    std::vector<uint32_t> h_pnt_len_vec;

    //#polygon indices for those contain points in h_pnt_idx_vec; sequentially concatenated
    std::vector<uint32_t> h_poly_idx_vec;


    hipStream_t stream=0;
    rmm::mr::device_memory_resource* mr=rmm::mr::get_default_resource();

    SBBox<double> setup_polygons(const char *file_name,uint8_t type)
    {
        std::vector<int> g_len_v,f_len_v,r_len_v;
        std::vector<double> x_v, y_v;
        GDALAllRegister();
        GDALDatasetH hDS = GDALOpenEx(file_name, GDAL_OF_VECTOR, nullptr, nullptr, nullptr );
        if(hDS==nullptr)
        {
            std::cout<<"Failed to open ESRI Shapefile dataset "<< file_name<<std::endl;
            exit(-1);
        }
        //a shapefile abstracted as a GDALDatasetGetLayer typically has only one layer
        OGRLayerH hLayer = GDALDatasetGetLayer( hDS,0 );

        this->h_ogr_polygon_vec.clear();
        this->h_geos_polygon_vec.clear();
        this->h_org_poly_idx_vec.clear();
        
        //type: 0 for all, 1 for simple polygons and 2 for multi-polygons
        uint32_t num_f=ReadLayer(hLayer,g_len_v,f_len_v,r_len_v,x_v,y_v,type,h_ogr_polygon_vec,h_org_poly_idx_vec);
        assert(num_f>0);
        
        h_geos_polygon_vec.clear();
        GEOSContextHandle_t hGEOSCtxt = OGRGeometry::createGEOSContext();
        for(uint32_t i=0;i<num_f;i++)
        {
            OGRGeometry *poOGRPoly=h_ogr_polygon_vec[i];
            GEOSGeometry *poGEOSPoly = poOGRPoly->exportToGEOS(hGEOSCtxt);
            h_geos_polygon_vec.push_back(poGEOSPoly);      	
        }

        //num_group=g_len_v.size();
        this->num_poly=f_len_v.size();
        this->num_ring=r_len_v.size();
        this->num_vertex=x_v.size();

        uint32_t *h_poly_flen=new uint32_t[num_poly];
        uint32_t *h_poly_rlen=new uint32_t[num_ring];
        assert(h_poly_flen!=nullptr && h_poly_rlen!=nullptr);
        
        this->h_poly_x=new double [num_vertex];
        this->h_poly_y=new double [num_vertex];
        assert(h_poly_x!=nullptr && h_poly_y!=nullptr);

        std::copy_n(f_len_v.begin(),num_poly,h_poly_flen);
        std::copy_n(r_len_v.begin(),num_ring,h_poly_rlen);
        std::copy_n(x_v.begin(),num_vertex,h_poly_x);
        std::copy_n(y_v.begin(),num_vertex,h_poly_y);
        std::cout<<"setup_polygons: num_poly="<<num_poly<<" num_ring="<<num_ring<<" num_vertex="<<num_vertex<<std::endl;

        //note that the bbox of all polygons will used as the Area of Intersects (AOI) to join points with polygons 
        double x1=*(std::min_element(x_v.begin(),x_v.end()));
        double x2=*(std::max_element(x_v.begin(),x_v.end()));
        double y1=*(std::min_element(y_v.begin(),y_v.end()));
        double y2=*(std::max_element(y_v.begin(),y_v.end()));
        std::cout<<"read_polygon_bbox: x_min="<<x1<<"  y_min="<<y1<<" x_max="<<x2<<" y_max="<<y2<<std::endl;

        return SBBox<double>(thrust::make_tuple(x1,y1), thrust::make_tuple(x2,y2));
    }

    void compare_random_points(uint32_t num_samples,uint32_t num_print_interval,bool using_geos)
    {
        std::cout<<"compare_random_points: num_quadrants="<<this->num_quadrants
            <<" num_pp_pair="<<this->num_pp_pairs<<" num_samples="<<num_samples<<std::endl;
        
        std::vector<uint32_t> rand_indices;
        gen_rand_idx(rand_indices,this->num_pnts,num_samples);

        timeval t0,t1;
        gettimeofday(&t0, nullptr);

        //h_pnt_idx_vec, h_pnt_len_vec and h_poly_idx_vec will be cleared first
  
        if(using_geos)
        {
            rand_points_geos_pip_test(num_print_interval,rand_indices, this->h_geos_polygon_vec,this->h_pnt_idx_vec,
                this->h_pnt_len_vec,this->h_poly_idx_vec,this->h_pnt_x,this->h_pnt_y);
        }
        else
        {
            rand_points_ogr_pip_test(num_print_interval,rand_indices, this->h_ogr_polygon_vec,this->h_pnt_idx_vec,
                this->h_pnt_len_vec,this->h_poly_idx_vec,this->h_pnt_x,this->h_pnt_y);
         }       
        gettimeofday(&t1, nullptr);
        float cpu_time=cuspatial::calc_time("cpu random sampling computing time = ",t0,t1);
    }
  
    void compare_matched_pairs(uint32_t num_samples,uint32_t num_print_interval,bool using_geos)
    {
        std::cout<<"compare_matched_pairs: num_quadrants="<<this->num_quadrants<<" num_pq_pairs"<<this->num_pq_pairs
            <<" num_pp_pair="<<this->num_pp_pairs<<" num_samples="<<num_samples<<std::endl;

        std::vector<uint32_t> rand_indices;
        gen_rand_idx(rand_indices,this->num_pq_pairs,num_samples);

        timeval t0,t1;
        gettimeofday(&t0, nullptr);
        
        if(using_geos)
        {
            matched_pairs_geos_pip_test(num_print_interval,rand_indices,
                this->h_pq_quad_idx,this->h_pq_poly_idx,this->h_qt_length,this->h_qt_fpos,
                this->h_geos_polygon_vec,this->h_pnt_idx_vec,this->h_pnt_len_vec,this->h_poly_idx_vec,
                this->h_pnt_x,this->h_pnt_y);
        }
        else
        {
            matched_pairs_ogr_pip_test(num_print_interval,rand_indices,
                this->h_pq_quad_idx,this->h_pq_poly_idx,this->h_qt_length,this->h_qt_fpos,
                this->h_ogr_polygon_vec,this->h_pnt_idx_vec,this->h_pnt_len_vec,this->h_poly_idx_vec,
                this->h_pnt_x,this->h_pnt_y);
   
        }
        gettimeofday(&t1, nullptr);
        float cpu_time=cuspatial::calc_time("cpu matched-pair computing time",t0,t1);                
    }

    void read_nyc_taxi(const char *file_name)
    {
        CUDF_EXPECTS(file_name!=NULL,"file_name can not be NULL");
        FILE *fp=fopen(file_name,"rb");
        CUDF_EXPECTS(fp!=NULL, "can not open file for input");
        CUDF_EXPECTS(fread(&(this->num_pnts),sizeof(uint32_t),1,fp)==1,"reading num_pnt failed");
        CUDF_EXPECTS(fread(&(this->num_quadrants),sizeof(uint32_t),1,fp)==1,"reading num_quadrants failed");
        CUDF_EXPECTS(fread(&(this->num_pq_pairs),sizeof(uint32_t),1,fp)==1,"reading num_pq_pairs failed");
        CUDF_EXPECTS(fread(&(this->num_pp_pairs),sizeof(uint32_t),1,fp)==1,"reading num_pp_pairs failed");
        std::cout<<"num_pnts="<<num_pnts<<" num_quadrants="<<num_quadrants<<" num_pq_pairs="<<num_pq_pairs<<" num_pp_pairs="<<num_pp_pairs<<std::endl;
    
        std::cout<<"reading points..."<<std::endl;
        this->h_pnt_x=new double[this->num_pnts];
        this->h_pnt_y=new double[this->num_pnts];
        CUDF_EXPECTS( this->h_pnt_x!=NULL && this->h_pnt_y!=NULL,"allocating memory for points on host failed");
    
        CUDF_EXPECTS(fread(this->h_pnt_x,sizeof(double),this->num_pnts,fp)==this->num_pnts,"reading h_pnt_x failed");
        CUDF_EXPECTS(fread(this->h_pnt_y,sizeof(double),this->num_pnts,fp)==this->num_pnts,"reading h_pnt_y failed");
        
        std::cout<<"reading quadrants..."<<std::endl;
        this->h_qt_length=new uint32_t[this->num_quadrants];
        this->h_qt_fpos=new uint32_t[this->num_quadrants];
        CUDF_EXPECTS( this->h_qt_length!=NULL && this->h_qt_fpos!=NULL,"allocating memory for quadrants on host failed");
  
        CUDF_EXPECTS(fread(this->h_qt_length,sizeof(uint32_t),this->num_quadrants,fp)==this->num_quadrants,"reading h_qt_length failed");
        CUDF_EXPECTS(fread(this->h_qt_fpos,sizeof(uint32_t),this->num_quadrants,fp)==this->num_quadrants,"reading h_qt_fpos failed");

        std::cout<<"reading quadrant/polygon pairs..."<<std::endl;
        this->h_pq_quad_idx=new uint32_t[this->num_pq_pairs];    
        this->h_pq_poly_idx=new uint32_t[this->num_pq_pairs];
        CUDF_EXPECTS( this->h_pq_poly_idx!=NULL && this->h_pq_quad_idx!=NULL,"allocating memory for quadrant-polygon pairs on host failed");

        CUDF_EXPECTS(fread(this->h_pq_quad_idx,sizeof(uint32_t),this->num_pq_pairs,fp)==this->num_pq_pairs,"reading h_pq_quad_idx failed");
        CUDF_EXPECTS(fread(this->h_pq_poly_idx,sizeof(uint32_t),this->num_pq_pairs,fp)==this->num_pq_pairs,"reading h_pq_poly_idx failed");

        std::cout<<"reading point/polygon pairs..."<<std::endl;
        this->h_pp_poly_idx=new uint32_t[this->num_pp_pairs];
        this->h_pp_pnt_idx=new uint32_t[this->num_pp_pairs];    
        CUDF_EXPECTS(this->h_pp_poly_idx!=NULL && this->h_pp_pnt_idx!=NULL,"allocating memory for point-polygon pairs on host failed");

        CUDF_EXPECTS(fread(this->h_pp_poly_idx,sizeof(uint32_t),this->num_pp_pairs,fp)==this->num_pp_pairs,"reading h_pp_poly_idx failed");
        CUDF_EXPECTS(fread(this->h_pp_pnt_idx,sizeof(uint32_t),this->num_pp_pairs,fp)==this->num_pp_pairs,"reading h_pp_pnt_idx failed");   

if(0)
{
        for(uint32_t i=0;i<this->num_pp_pairs;i++)
        {
            if(i%100==0)
                std::cout<<i<<" "<<h_pp_poly_idx[i]<<" "<<h_pp_pnt_idx[i]<<std::endl;
        }
}

    }

    void tear_down()
    {
        delete[] this->h_poly_x; this->h_poly_x=nullptr;
        delete[] this->h_poly_y; this->h_poly_y=nullptr;

        delete[] this->h_pnt_x; this->h_pnt_x=nullptr;
        delete[] h_pnt_y; h_pnt_y=nullptr;
        
        delete[] this->h_pq_quad_idx; this->h_pq_quad_idx=nullptr;
        delete[] h_pq_poly_idx; h_pq_poly_idx=nullptr;
        
        delete[] this->h_qt_length; this->h_qt_length=nullptr;
        delete[] this->h_qt_fpos; this->h_qt_fpos=nullptr;
    }

};

/* 
 * There could be multple configureations (minior ones are inside parentheses): 
 * pick one of three polygon datasets
 * choose from compare_random_points and compare_matched_pairs 
*/

TEST_F(SpatialJoinNYCTaxiVerify, verify)
{
    const char* env_p = std::getenv("CUSPATIAL_DATA");
    CUDF_EXPECTS(env_p!=nullptr,"CUSPATIAL_DATA environmental variable must be set");
   
    //#0: NYC taxi zone: 263 polygons
    //from https://s3.amazonaws.com/nyc-tlc/misc/taxi_zones.zip
    //#1: NYC Community Districts: 71 polygons
    //from https://www1.nyc.gov/assets/planning/download/zip/data-maps/open-data/nycd_11aav.zip
    //#2: NYC Census Tract 2000 data: 2216 polygons
    //from: https://www1.nyc.gov/assets/planning/download/zip/data-maps/open-data/nyct2000_11aav.zip
 
    //note that the polygons and the points need to use the same projection 
    //all the three polygon datasets use epsg:2263 (unit is foot) for NYC/Long Island area 

    enum POLYID {taxizone_id=0,cd_id,ct_id};    
    POLYID sel_id=taxizone_id;

    const char * shape_files[]={"taxi_zones.shp","nycd_11a_av/nycd.shp","nyct2000_11a_av/nyct2000.shp"};
    
    const char * bin_files[]={"nyc_taxizone_2009_1.bin","nyc_cd_2009_12.bin","nyc_ct_2009_12.bin"};
    
    read_nyc_taxi(bin_files[sel_id]);

    std::cout<<"loading NYC polygon data..........."<<std::endl;

    std::string shape_filename=std::string(env_p)+std::string(shape_files[sel_id]); 
    
    std::cout<<"Using shapefile "<<shape_filename<<std::endl;

    //uint8_t poly_type=2; //multi-polygons only 
    //uint8_t poly_type=1; //single-polygons only 
    uint8_t poly_type=0; //all polygons

    this->setup_polygons(shape_filename.c_str(),poly_type);

    std::cout<<"running GDAL/OGR or GEOS CPU code for comparison/verification..........."<<std::endl;

    uint32_t num_print_interval=100;
    
    bool using_geos=false;

    //type 1: random points
    //uint32_t num_pnt_samples=this->num_pnts;
    uint32_t num_pnt_samples=10000;
    this->compare_random_points(num_pnt_samples,num_print_interval,using_geos);

    //type 2: random quadrant/polygon pairs
    //uint32_t num_quad_samples=10000;
    //this->compare_matched_pairs(num_quad_samples,num_print_interval,using_geos);

    //for unknown reason, the following two lines can not be compiled in spatial_join_test_utility.cu
    //h_pnt_search_idx and h_poly_search_idx do not need to be freed as the destructor of std::vector does it
    uint32_t * h_pnt_search_idx=&(h_pnt_idx_vec[0]);
    uint32_t * h_poly_search_idx=&(h_poly_idx_vec[0]);

    bool verified=compute_mismatch(this->num_pp_pairs,this->h_org_poly_idx_vec,
        h_pnt_search_idx,this->h_pnt_len_vec,h_poly_search_idx,
        this->h_pp_pnt_idx,this->h_pp_poly_idx,   
        this->h_pnt_x,this->h_pnt_y,mr,stream);
    std::string msg=verified ? "verified" : "mismatch";
    std::cout<<"comparison/verification result: " << msg << std::endl;
    this->tear_down();

}//TEST_F

